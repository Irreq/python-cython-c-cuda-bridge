#include "hip/hip_runtime.h"


#include <math.h>
#include <stdio.h>

// Thread block size
#define BLOCK_SIZE 256

// Expose function
extern "C" void cudaExposedWrapper(int *res, const int *first, const int *last, int n_bytes);

// Your kernel to be indirectly called from Python
__global__ void multiplyKernel(int *res, const int *a, const int *b, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        res[i] = a[i] * b[i];
    }
}

// Cuda Wrapper for `multiplyKernel` used by C or Cython code
void cudaExposedWrapper(int *res, const int *first, const int *last, int n_bytes)
{
    // Setup buffers for GPU
    int *dev_res = nullptr;
    int *dev_first = nullptr;
    int *dev_last = nullptr;

    // Allocate memory on GPU for three vectors
    hipMalloc((void **)&dev_res, n_bytes * sizeof(int));
    hipMalloc((void **)&dev_first, n_bytes * sizeof(int));
    hipMalloc((void **)&dev_last, n_bytes * sizeof(int));

    // Copy allocated host memory to device
    hipMemcpy(dev_first, first, n_bytes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_last, last, n_bytes * sizeof(int), hipMemcpyHostToDevice);

    // Compute the result using one thread per element in vector
    // 2 is number of computational blocks and (n_bytes + 1) / 2 is a number of threads in a block
    multiplyKernel<<<2, (n_bytes + 1) / 2>>>(dev_res, dev_first, dev_last, n_bytes);

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(res, dev_res, n_bytes * sizeof(int), hipMemcpyDeviceToHost);

    // Release allocated memory
    hipFree(dev_res);
    hipFree(dev_first);
    hipFree(dev_last);

    hipDeviceReset();
}